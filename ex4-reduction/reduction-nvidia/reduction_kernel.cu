#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
    Parallel reduction kernels
*/

#ifndef _REDUCE_KERNEL_H_
#define _REDUCE_KERNEL_H_

#define _CG_ABI_EXPERIMENTAL
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <stdio.h>

namespace cg = cooperative_groups;

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template <class T>
struct SharedMemory {
  __device__ inline operator T *() {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }

  __device__ inline operator const T *() const {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }
};

// specialize for double to avoid unaligned memory
// access compile errors
template <>
struct SharedMemory<double> {
  __device__ inline operator double *() {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }

  __device__ inline operator const double *() const {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }
};

template <class T>
__device__ __forceinline__ T warpReduceSum(unsigned int mask, T mySum) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    mySum += __shfl_down_sync(mask, mySum, offset);
  }
  return mySum;
}

#if __CUDA_ARCH__ >= 800
// Specialize warpReduceFunc for int inputs to use __reduce_add_sync intrinsic
// when on SM 8.0 or higher
template <>
__device__ __forceinline__ int warpReduceSum<int>(unsigned int mask,
                                                  int mySum) {
  mySum = __reduce_add_sync(mask, mySum);
  return mySum;
}
#endif

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved
   inactivity means that no whole warps are active, which is also very
   inefficient */
template <class T>
__global__ void reduce0(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  T *sdata = SharedMemory<T>();

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (i < n) ? g_idata[i] : 0;

  cg::sync(cta);

  // do reduction in shared mem
  for (unsigned int s = 1; s < blockDim.x; s *= 2) {
    // modulo arithmetic is slow!
    if ((tid % (2 * s)) == 0) {
      sdata[tid] += sdata[tid + s];
    }

    cg::sync(cta);
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/* This version uses contiguous threads, but its interleaved
   addressing results in many shared memory bank conflicts.
*/
template <class T>
__global__ void reduce1(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  T *sdata = SharedMemory<T>();

  // load shared memcd re 
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (i < n) ? g_idata[i] : 0;

  cg::sync(cta);

  // do reduction in shared mem
  for (unsigned int s = 1; s < blockDim.x; s *= 2) {
    int index = 2 * s * tid;

    if (index < blockDim.x) {
      sdata[index] += sdata[index + s];
    }

    cg::sync(cta);
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/*
    This version uses sequential addressing -- no divergence or bank conflicts.
*/
template <class T>
__global__ void reduce2(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  T *sdata = SharedMemory<T>();

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (i < n) ? g_idata[i] : 0;

  cg::sync(cta);

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }

    cg::sync(cta);
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void reduce(int size, int threads, int blocks, int whichKernel, T *d_idata,
            T *d_odata) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize =
      (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

  // choose which of the optimized versions of reduction to launch
  switch (whichKernel) {
    case 0:
      reduce0<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
      break;

    case 1:
      reduce1<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
      break;

    case 2:
      reduce2<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
      break;    
        
    default:
      break;
  }
}

// Instantiate the reduction function for 3 types
template void reduce<int>(int size, int threads, int blocks, int whichKernel,
                          int *d_idata, int *d_odata);

template void reduce<float>(int size, int threads, int blocks, int whichKernel,
                            float *d_idata, float *d_odata);

template void reduce<double>(int size, int threads, int blocks, int whichKernel,
                             double *d_idata, double *d_odata);

#endif  // #ifndef _REDUCE_KERNEL_H_
