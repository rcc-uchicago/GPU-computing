/**
 * Vector scale: x <= x * k
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
//#include <cuda_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Multiplies all the elements of vector x by a scalar value
 */
__global__ void scale(float *x, const float k, const int N) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  if (tid < N) {
    x[tid] = x[tid] * k;
  }
}

/**
 * Host main routine
 */
int main(void) {

  float k = 10.0;

  // Print the vector length to be used, and compute its size
  int numElements = 50000;
  size_t size = numElements * sizeof(float);
  printf("Vector scaling of %d elements: k = %f\n", numElements, k);

  // Allocate the host input vector x
  float *h_x = (float *)malloc(size);

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    h_x[i] = i;
  }

  // Allocate the device input vector x
  float *d_x = NULL;
  hipMalloc((void **)&d_x, size);

  // Copy the host input vector x in host memory to the device input
  // vectors in device memory
  
  hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
  scale<<<blocksPerGrid, threadsPerBlock>>>(d_x, k, numElements);

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  hipMemcpy(h_x, d_x, size, hipMemcpyDeviceToHost);

  // Verify that the result vector is correct
  for (int i = 0; i < numElements; ++i) {
    if (fabs(h_x[i]  - k * i) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");

  // Free device global memory
  hipFree(d_x);

  // Free host memory
  free(h_x);

  printf("Done\n");
  return 0;
}
